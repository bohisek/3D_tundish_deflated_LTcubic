#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : 3D_tundish.cu
 Author      : Jan Bohacek
 Version     :
 Copyright   : 
 Description : laminar flow in three-dimensional tundish in continuous casting
 ============================================================================
 */


#include <iostream>
//#include <stdio.h>
//#include <algorithm>
//#include <numeric>
#include <fstream>
#include <sstream>
#include <cstring>
//#include <ctime>
#include <math.h>
#include <iomanip>

#define DEFL
#define DEFLDIR // if commented solveICCG()
#define MIC0  // with DEFLDIR, if commented IC0 (incomplete Cholesky zero fill)

using namespace std;

typedef double T;	// precision of calculation

typedef struct {
	int Nx; 		// x-coordinate
	int Ny;			// y
	int Nz;			// z
	T dx; 			// dx = dy = dz
} Dimensions;		// dimensions of geometry

typedef struct {
	int steps;		// number of timesteps (-)
	int maxIterSIMPLE; // maximum number of SIMPLE iterations
	T CFL;			// Courant number
	T dt;			// timestep size
	T UZ;			// inlet velocity
	T ac;			// volume of cell divided by timestep
	T blocks;		// for dot product
	T blockSize;	// -||-
	T maxResU;		// stopping criterion for velocity calculation
	T maxResP;		// 					      pressure
	T maxResSIMPLE; //						  SIMPLE 
	T urfU;			// under-relaxation factor U
	T urfP;			// 						   P
} Parameters;		// simulation settings


typedef struct {			// deflation
	unsigned int NxZ;
	unsigned int NyZ;
	unsigned int nDV;		// number of deflation vectors
	unsigned int nRowsZ;	// number of rows/columns for one deflation vector
	T maxresZ;
} ParametersZ;

typedef struct {
	T nu; 			// kinematic viscosity (m2/s)
	T rho;			// density
	T cp;			// specific heat
	T k;			// thermal conductivity
	T alpha;    	// thermal diffusivity (m2/s)
	T beta; 		// thermal expansion coefficient
} MaterialProperties;

// declare CPU fields
Dimensions  dims;
Parameters  params;
ParametersZ paramsZ;
MaterialProperties liquid;

// cache constant GPU fields
__constant__ Dimensions d_dims;
__constant__ Parameters d_params;
__constant__ ParametersZ d_paramsZ;
__constant__ MaterialProperties d_liquid;
__constant__ int d_A[64][64];	// coefficient matrix for tricubic interpolation

#include "cpuFunctions.h"
#include "cudaFunctions.h"
#include "cpuFunctionsDeflation.h"
#include "cudaFunctionsDeflation.h"

int main()
{
	
		
	cout << "--flow in 3D tundish---" << endl;
		
	// geometry
	dims.Nx = 256;
	dims.Ny = 64;
	dims.Nz = 64;
	dims.dx = 0.001;
	
	// parameters deflation 
	paramsZ.nRowsZ = 16; 
	paramsZ.NxZ = dims.Nx/paramsZ.nRowsZ; 	// number of course cells in X 
	paramsZ.NyZ = dims.Ny/paramsZ.nRowsZ; 	// number of course cells in Y 
	paramsZ.nDV = paramsZ.NxZ * paramsZ.NyZ * dims.Nz/paramsZ.nRowsZ;		// size of coarse system
	paramsZ.maxresZ  = 1e-8;
	
	// paramaters
	params.steps     = 5000;
	params.CFL       = 0.5;
	params.UZ        = -0.5;
	params.dt        = params.CFL * dims.dx / fabs(params.UZ);
	params.ac        = dims.dx*dims.dx/params.dt;
	params.blocks    = 256;  
	params.blockSize = 128;  
	params.maxResU       = 1e-3;
	params.maxResP       = 1e-3;
	params.maxResSIMPLE  = 1e-3;
	params.maxIterSIMPLE = 1;
	params.urfU          = 0.7;
	params.urfP          = 0.3;
	params.maxIterSIMPLE = 20;
	
	// material properties
	liquid.nu  = 0.000001;   // water 1e-6 m2/s
	liquid.rho = 1000;
	
	cout << "For Courant number of " << params.CFL << " the timestep size is " << params.dt << endl;
	
	// CPU fields
	T *ux;		// ux-component of velocity
	T *uy;		// uy
	T *uz;		// uy
	T *p;		// pressure
	T *m;		// mass balance
	T *hrh,*hsg;	// dot products
	T rhNew, rhOld, sg, ap, bt;
	T endIter, endIterP, rhNewSIMPLE;
	int iter, iterSIMPLE;
		
#ifdef DEFL
	// CPU fields deflation
	T *pc,*pf,*ps,*pw;
	T *pzc, *pzf, *pzw, *pzs;
	T *ec, *ef, *es, *ew;
	T *hrZ, *hyZ, *hqZ, *hpZ, *hsZ;
	T *L;
	T *lc,*lf,*ls,*lw;
	
	// GPU fields deflation
	T *dpzc, *dpzf, *dpzw, *dpzs;
    T *dec, *def, *des, *dew;
    T *drZ, *dyZ, *dqZ, *dpZ, *dsZ;
    T *drhs;
#endif
		
	// GPU fields
	T *dux , *duy , *duz;		// velocity components
	T *duxhalf, *duyhalf, *duzhalf;
	T *duxo, *duyo, *duzo;		// old values
	T *dp, *dpo;				// pressure and old value
	T *dm;      				// mass balance 
	T *duxtemp, *duytemp, *duztemp;								// pointers for swapping fields
	T *duxc,*duxf,*duxs,*duxw,*dkuxc,*dkuxf,*dkuxs,*dkuxw; 		// Aux
	T *drx,*dqx,*dzx,*dpx;										// Aux
	T *duyc,*duyf,*duys,*duyw, *dkuyc,*dkuyf,*dkuys,*dkuyw; 	// Auy
	T *dry,*dqy,*dzy,*dpy;										// Auy
	T *duzc,*duzf,*duzs,*duzw, *dkuzc,*dkuzf,*dkuzs,*dkuzw; 	// Auz
	T *drz,*dqz,*dzz,*dpz;										// Auz
	T *dpc,*dpf,*dps,*dpw,*dkpc,*dkpf,*dkps,*dkpw; 				// Ap
	T *drp,*dqp,*dzp,*dpp;										// Ap
	T *drh,*dsg;												// dot products
	T *duxdx, *duxdy, *duxdz, *duxdxdy, *duxdxdz, *duxdydz, *duxdxdydz;		// derivatives fot tricubic interpolation
	T *duydx, *duydy, *duydz, *duydxdy, *duydxdz, *duydydz, *duydxdydz;
	T *duzdx, *duzdy, *duzdz, *duzdxdy, *duzdxdz, *duzdydz, *duzdxdydz;
	T *x_ux, *y_ux, *z_ux, *x_uy, *y_uy, *z_uy, *x_uz, *y_uz, *z_uz; 
	
	
	// GPU parameters
	int THREADS_PER_BLOCK = 1024;
	int BLOCKS = ((dims.Nx+2)*(dims.Ny+2)*(dims.Nz+2)+THREADS_PER_BLOCK-1) / THREADS_PER_BLOCK;	// larger in order to have BLOCKS*THREADS_PER_BLOCK > Nx*Ny*Nz
	dim3 dimBlockZ(paramsZ.nRowsZ,paramsZ.nRowsZ,1);
	
	int THREADS_PER_BLOCK_NEW = 32;
	int BLOCKS_NEW = ((dims.Nx+2)*(dims.Ny+2)*(dims.Nz+2)+THREADS_PER_BLOCK_NEW-1) / THREADS_PER_BLOCK_NEW;
	// taken from CUDA by example
			
	// initialize fields 
	cpuInit(ux, uy, uz, p, m, hrh, hsg);
	cudaInit(dux, duy, duz, dp, dm, duxo, duyo, duzo, dpo,
			 duxhalf, duyhalf, duzhalf,
			 duxc, duxf, duxs, duxw, dkuxc, dkuxf, dkuxs, dkuxw, 	// Aux
			 drx, dqx, dzx, dpx,									// Aux
			 duyc, duyf, duys, duyw, dkuyc, dkuyf, dkuys, dkuyw, 	// Auy
			 dry, dqy, dzy, dpy,									// Auy
			 duzc, duzf, duzs, duzw, dkuzc, dkuzf, dkuzs, dkuzw, 	// Auz
			 drz, dqz, dzz, dpz,									// Auz
			 dpc, dpf, dps, dpw, dkpc, dkpf, dkps, dkpw, 			// Ap
			 drp, dqp, dzp, dpp,									// Ap
			 drh, dsg);
	cudaInitTricubicDerivatives(duxdx, duxdy, duxdz,
				duxdxdy, duxdxdz, duxdydz, duxdxdydz,
				duydx, duydy, duydz,
				duydxdy, duydxdz, duydydz, duydxdydz,
				duzdx, duzdy, duzdz,
				duzdxdy, duzdxdz, duzdydz, duzdxdydz,
				x_ux, y_ux, z_ux,
				x_uy, y_uy, z_uy,
			    x_uz, y_uz, z_uz);
	
	// patch anything to dux
	//patchDux<<<BLOCKS,THREADS_PER_BLOCK>>>(dux);
	
	// patch anything to duy
	//patchDuy<<<BLOCKS,THREADS_PER_BLOCK>>>(duy);
	
	// patch anything to duz
	//patchDuz<<<BLOCKS,THREADS_PER_BLOCK>>>(duz);
	
	/*// copy back to host and save
	hipMemcpy(ux, dux, sizeof(T)*(dims.Nx+2)*(dims.Ny+2)*(dims.Nz+2), hipMemcpyDeviceToHost);
	hipMemcpy(uy, duy, sizeof(T)*(dims.Nx+2)*(dims.Ny+2)*(dims.Nz+2), hipMemcpyDeviceToHost);
	hipMemcpy(uz, duz, sizeof(T)*(dims.Nx+2)*(dims.Ny+2)*(dims.Nz+2), hipMemcpyDeviceToHost);
	hipMemcpy(p,   dp, sizeof(T)*dims.Nx    * dims.Ny   *(dims.Nz+2), hipMemcpyDeviceToHost);
	saveDataInTime(ux, uy, uz, p, m, (T)0, "testTundish");*/
	
	// Aux (x-component of velocity)
	Aux<<<BLOCKS,THREADS_PER_BLOCK>>>(duxc, duxf, duxs, duxw);
	// AuxInlet not necessary, velocity inlet condition ux=0 is the same as no slip condition at wall
	AuxOutlet<<<1,100>>>(duxc,200,15);
	makeTNS1<<<BLOCKS,THREADS_PER_BLOCK>>>(dkuxc,dkuxf,dkuxs,dkuxw,duxc,duxf,duxs,duxw,dims.Nx-1,dims.Ny,dims.Nz);
	
	// Auy (y-component of velocity)
	Auy<<<BLOCKS,THREADS_PER_BLOCK>>>(duyc, duyf, duys, duyw);
	//AuyInlet not necessary
	AuyOutlet<<<1,100>>>(duyc,200,15);
	makeTNS1<<<BLOCKS,THREADS_PER_BLOCK>>>(dkuyc,dkuyf,dkuys,dkuyw,duyc,duyf,duys,duyw,dims.Nx,dims.Ny-1,dims.Nz);
	
	// Auz (z-component of velocity)
	Auz<<<BLOCKS,THREADS_PER_BLOCK>>>(duzc, duzf, duzs, duzw);
	//AuzInlet not necessary
	AuzOutlet<<<1,100>>>(duzc,200,15);
	makeTNS1<<<BLOCKS,THREADS_PER_BLOCK>>>(dkuzc,dkuzf,dkuzs,dkuzw,duzc,duzf,duzs,duzw,dims.Nx,dims.Ny,dims.Nz-1);
		
	// Ap (pressure)
	Ap<<<BLOCKS,THREADS_PER_BLOCK>>>(dpc, dpf, dps, dpw);
	ApOutlet<<<1,100>>>(dpc,200,15); // Dirichlet, p=0
	makeTNS1<<<BLOCKS,THREADS_PER_BLOCK>>>(dkpc,dkpf,dkps,dkpw,dpc,dpf,dps,dpw,dims.Nx,dims.Ny,dims.Nz);
	
#ifdef DEFL
	cpuInitDeflation(pzc, pzf, pzs, pzw,
				ec, ef, es, ew,
				pc, pf, ps, pw,
				lc, lf, ls, lw,
				hrZ,hyZ,hqZ,hpZ,hsZ,
				L);
	hipMemcpy(pc,dpc,sizeof(T)*(dims.Nx*dims.Ny*dims.Nz+2*dims.Nx*dims.Ny),hipMemcpyDeviceToHost);
	hipMemcpy(pf,dpf,sizeof(T)*(dims.Nx*dims.Ny*dims.Nz+dims.Nx*dims.Ny  ),hipMemcpyDeviceToHost);
	hipMemcpy(ps,dps,sizeof(T)*(dims.Nx*dims.Ny*dims.Nz+dims.Nx     	  ),hipMemcpyDeviceToHost);
	hipMemcpy(pw,dpw,sizeof(T)*(dims.Nx*dims.Ny*dims.Nz+1                ),hipMemcpyDeviceToHost);
	initAZ(pzc,pzf,pzs,pzw,pc,pf,ps,pw);
	initE(ec,ef,es,ew,pc,pf,ps,pw);
	cudaInitDeflation(dpzc,dpzf,dpzs,dpzw,
				dec,def,des,dew,
				drZ,dyZ,dqZ,dpZ,
				drhs,
				ec,ef,es,ew,
				pzc,pzf,pzs,pzw);
#ifdef DEFLDIR
	Chol(L,ec,ef,es,ew); // Cholesky factorization
#else
	IChol(lc,lf,ls,lw,ec,ef,es,ew); // incomplete Cholesky factorization with zero fill (IC(0) or MIC(0))
#endif
#endif
	
	/*for (int i=0; i<paramsZ.nDV;i++){
		//cout << ec[i+paramsZ.NxZ*paramsZ.NyZ] << endl;
		cout << ew[i] << endl;
	}
	*/
	
	
	hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	
	
	for (int miter=0; miter<params.steps; miter++) {
	
		// boundary conditions
		bcVelWallNoslip<<<BLOCKS,THREADS_PER_BLOCK>>>(dux, duy, duz);	// no slip at walls
		bcVelInlet<<<1,100>>>(duz, params.UZ, 50, 45);					// bcVelInlet<<<1,inletWidth>>>(dux, duy, duz, velocity, first index in x, first index in y);
		bcVelOutlet<<<1,100>>>(dux, duy, duz, 200, 15);					// bcVelOutlet<<<1,outletwidth>>>(dux, duy, duz, first index in x, first index in y);
		
		// Temperton & Staniforth 3/2u(t) - 1/2u(t-1)
		hipMemcpy(duxhalf, dux, sizeof(T)*(dims.Nx+2)*(dims.Ny+2)*(dims.Nz+2), hipMemcpyDeviceToDevice);
		hipMemcpy(duyhalf, duy, sizeof(T)*(dims.Nx+2)*(dims.Ny+2)*(dims.Nz+2), hipMemcpyDeviceToDevice);
		hipMemcpy(duzhalf, duz, sizeof(T)*(dims.Nx+2)*(dims.Ny+2)*(dims.Nz+2), hipMemcpyDeviceToDevice);
		AXPY<<<BLOCKS,THREADS_PER_BLOCK>>>(duxhalf,duxo,(T)-0.5,(T)1.5,dims.Nx+2,dims.Ny+2,dims.Nz+2);
		AXPY<<<BLOCKS,THREADS_PER_BLOCK>>>(duyhalf,duyo,(T)-0.5,(T)1.5,dims.Nx+2,dims.Ny+2,dims.Nz+2);
		AXPY<<<BLOCKS,THREADS_PER_BLOCK>>>(duzhalf,duzo,(T)-0.5,(T)1.5,dims.Nx+2,dims.Ny+2,dims.Nz+2);
		
		//swap old and new arrays for next timestep
		duxtemp = duxo; duxo = dux; dux = duxtemp;
		duytemp = duyo; duyo = duy; duy = duytemp;
		duztemp = duzo; duzo = duz; duz = duztemp;
		
		// get derivatives for tricubic interpolations
		getTricubicDerivatives<<<BLOCKS_NEW,THREADS_PER_BLOCK_NEW>>>(duxdx, duxdy, duxdz, duxdxdy, duxdxdz, duxdydz, duxdxdydz,
																	 duydx, duydy, duydz, duydxdy, duydxdz, duydydz, duydxdydz,
																	 duzdx, duzdy, duzdz, duzdxdy, duzdxdz, duzdydz, duzdxdydz,
																	 duxhalf, duyhalf, duzhalf);
		//get departure points of Lagrangian trajectories (Temperton & Staniforth)
		advectUxDeparturePoint<<<BLOCKS_NEW,THREADS_PER_BLOCK_NEW>>>(x_ux, y_ux, z_ux, duxhalf, duyhalf, duzhalf,
													   duxdx, duxdy, duxdz, duxdxdy, duxdxdz, duxdydz, duxdxdydz,
													   duydx, duydy, duydz, duydxdy, duydxdz, duydydz, duydxdydz,
													   duzdx, duzdy, duzdz, duzdxdy, duzdxdz, duzdydz, duzdxdydz);
		advectUyDeparturePoint<<<BLOCKS_NEW,THREADS_PER_BLOCK_NEW>>>(x_uy, y_uy, z_uy, duxhalf, duyhalf, duzhalf,
													   duxdx, duxdy, duxdz, duxdxdy, duxdxdz, duxdydz, duxdxdydz,
													   duydx, duydy, duydz, duydxdy, duydxdz, duydydz, duydxdydz,
													   duzdx, duzdy, duzdz, duzdxdy, duzdxdz, duzdydz, duzdxdydz);
		advectUzDeparturePoint<<<BLOCKS_NEW,THREADS_PER_BLOCK_NEW>>>(x_uz, y_uz, z_uz, duxhalf, duyhalf, duzhalf,
													   duxdx, duxdy, duxdz, duxdxdy, duxdxdz, duxdydz, duxdxdydz,
													   duydx, duydy, duydz, duydxdy, duydxdz, duydydz, duydxdydz,
													   duzdx, duzdy, duzdz, duzdxdy, duzdxdz, duzdydz, duzdxdydz);
		// calculate new derivatives for tricubic interpolation using uxo, uyo, uzo
		getTricubicDerivatives<<<BLOCKS_NEW,THREADS_PER_BLOCK_NEW>>>(duxdx, duxdy, duxdz, duxdxdy, duxdxdz, duxdydz, duxdxdydz,
																	 duydx, duydy, duydz, duydxdy, duydxdz, duydydz, duydxdydz,
																	 duzdx, duzdy, duzdz, duzdxdy, duzdxdz, duzdydz, duzdxdydz,
																	 duxo, duyo, duzo);
		//advect horizontal and vertical velocity components (Temperton & Staniforth)
		advectUx<<<BLOCKS_NEW,THREADS_PER_BLOCK_NEW>>>(dux, x_ux, y_ux, z_ux, duxo,
													   duxdx, duxdy, duxdz, duxdxdy, duxdxdz, duxdydz, duxdxdydz,
													   duydx, duydy, duydz, duydxdy, duydxdz, duydydz, duydxdydz,
													   duzdx, duzdy, duzdz, duzdxdy, duzdxdz, duzdydz, duzdxdydz);
		advectUy<<<BLOCKS_NEW,THREADS_PER_BLOCK_NEW>>>(duy, x_uy, y_uy, z_uy, duyo,
													   duxdx, duxdy, duxdz, duxdxdy, duxdxdz, duxdydz, duxdxdydz,
													   duydx, duydy, duydz, duydxdy, duydxdz, duydydz, duydxdydz,
													   duzdx, duzdy, duzdz, duzdxdy, duzdxdz, duzdydz, duzdxdydz);
		advectUz<<<BLOCKS_NEW,THREADS_PER_BLOCK_NEW>>>(duz, x_uz, y_uz, z_uz, duzo,
													   duxdx, duxdy, duxdz, duxdxdy, duxdxdz, duxdydz, duxdxdydz,
													   duydx, duydy, duydz, duydxdy, duydxdz, duydydz, duydxdydz,
													   duzdx, duzdy, duzdz, duzdxdy, duzdxdz, duzdydz, duzdxdydz);
		
		bcVelWallNoslip<<<BLOCKS,THREADS_PER_BLOCK>>>(dux, duy, duz);	// no slip at walls
		bcVelInlet<<<1,100>>>(duz, params.UZ, 50, 45);					// bcVelInlet<<<1,inletWidth>>>(dux, duy, duz, velocity, first index in x, first index in y);
		bcVelOutlet<<<1,100>>>(dux, duy, duz, 200, 15);					// bcVelOutlet<<<1,outletwidth>>>(dux, duy, duz, first index in x, first index in y);
		
		hipMemcpy(duxo, dux, sizeof(T)*(dims.Nx+2)*(dims.Ny+2)*(dims.Nz+2), hipMemcpyDeviceToDevice);
		hipMemcpy(duyo, duy, sizeof(T)*(dims.Nx+2)*(dims.Ny+2)*(dims.Nz+2), hipMemcpyDeviceToDevice);
		hipMemcpy(duzo, duz, sizeof(T)*(dims.Nx+2)*(dims.Ny+2)*(dims.Nz+2), hipMemcpyDeviceToDevice);
		
		// ************ BEGIN SIMPLE **********
		iterSIMPLE    = 0;
		rhNewSIMPLE   = 1;
		
		/*// copy back to host and save
		hipMemcpy(ux, dux, sizeof(T)*(dims.Nx+2)*(dims.Ny+2)*(dims.Nz+2), hipMemcpyDeviceToHost);
		hipMemcpy(uy, duy, sizeof(T)*(dims.Nx+2)*(dims.Ny+2)*(dims.Nz+2), hipMemcpyDeviceToHost);
		hipMemcpy(uz, duz, sizeof(T)*(dims.Nx+2)*(dims.Ny+2)*(dims.Nz+2), hipMemcpyDeviceToHost);
		hipMemcpy(p,   dp, sizeof(T)*dims.Nx    * dims.Ny   *(dims.Nz+2), hipMemcpyDeviceToHost);
		saveDataInTime(ux, uy, uz, p, m, (T)0, "testTundish");*/
		
		
		while (rhNewSIMPLE > params.maxResSIMPLE) {   //(iterSIMPLE < params.maxIterSIMPLE) { 
		
			iterSIMPLE++;
		
			// ********** BEGIN solve UX **********
			duToDr<<<BLOCKS,THREADS_PER_BLOCK>>>(drx, dux,dims.Nx-1,dims.Ny,dims.Nz);						// drx := dux
			SpMV<<<BLOCKS,THREADS_PER_BLOCK>>>(dqx,duxc,duxf,duxs,duxw,drx,dims.Nx-1,dims.Ny,dims.Nz);		// q := Aux ux 
			//duToDr<<<BLOCKS,THREADS_PER_BLOCK>>>(drx, duxhalf,dims.Nx-1,dims.Ny,dims.Nz);
			duToDr<<<BLOCKS,THREADS_PER_BLOCK>>>(drx, duxo,dims.Nx-1,dims.Ny,dims.Nz);
			b<<<BLOCKS,THREADS_PER_BLOCK>>>(drx,dims.Nx-1,dims.Ny,dims.Nz);									// drx := bx
			// bxInlet not necessary as ux=0 there
			expMux<<<BLOCKS,THREADS_PER_BLOCK>>>(drx,duxo,dims.Nx-1,dims.Ny,dims.Nz); // explicit part of diffusion term (Crank-Nicolson scheme)			
			bpx<<<BLOCKS,THREADS_PER_BLOCK>>>(drx,dpo,dims.Nx-1,dims.Ny,dims.Nz);	    					// add grad(p) to rhs of Ax=b
			AXPY<<<BLOCKS,THREADS_PER_BLOCK>>>(drx,dqx,(T)-1.,(T)1.,dims.Nx-1,dims.Ny, dims.Nz); 	  		// r = r - q
			SpMV<<<BLOCKS,THREADS_PER_BLOCK>>>(dzx,dkuxc,dkuxf,dkuxs,dkuxw,drx,dims.Nx-1,dims.Ny,dims.Nz);	// z = M^(-1)r
			DOTGPU<T,128><<<params.blocks,params.blockSize,params.blockSize*sizeof(T)>>>(drh,drx,dzx,dims.Nx-1,dims.Ny,dims.Nz);
			hipMemcpy(hrh, drh, params.blocks*sizeof(T), hipMemcpyDeviceToHost);
			rhNew = dot(hrh,params.blocks);
			//cout << "Ux residual at start: " << rhNew << endl;
			endIter = rhNew * params.maxResU * params.maxResU;
			iter = 0;
			
			while (rhNew > endIter) {
				iter++;
				if (iter==1) {
					hipMemcpy(dpx, dzx, sizeof(T)*(dims.Nx-1)*dims.Ny*(dims.Nz+2),hipMemcpyDeviceToDevice);
				}
				else {
					bt = rhNew/rhOld;
					AXPY<<<BLOCKS,THREADS_PER_BLOCK>>>(dpx,dzx,(T)1.,bt,dims.Nx-1,dims.Ny,dims.Nz);   			// p = z + beta*p	
				}
				SpMV<<<BLOCKS,THREADS_PER_BLOCK>>>(dqx,duxc,duxf,duxs,duxw,dpx,dims.Nx-1,dims.Ny,dims.Nz);		// q := Aux p
				DOTGPU<T,128><<<params.blocks,params.blockSize,params.blockSize*sizeof(T)>>>(dsg, dpx, dqx, dims.Nx-1, dims.Ny,dims.Nz);
				hipMemcpy(hsg, dsg, params.blocks*sizeof(T), hipMemcpyDeviceToHost);
				sg = dot(hsg,params.blocks);
				ap = rhNew/sg;	// alpha = rhoNew / sigma
				AXPY<<<BLOCKS,THREADS_PER_BLOCK>>>(drx,dqx,-ap,(T)1.,dims.Nx-1,dims.Ny,dims.Nz); 				// r = r - alpha*q
				AXPY2<<<BLOCKS,THREADS_PER_BLOCK>>>(dux,dpx, ap,(T)1.,dims.Nx-1,dims.Ny,dims.Nz);  				// x = x + alpha*p; Note: sizeof(dux) != sizeof(dpx)
				SpMV<<<BLOCKS,THREADS_PER_BLOCK>>>(dzx,dkuxc,dkuxf,dkuxs,dkuxw,drx,dims.Nx-1,dims.Ny,dims.Nz);	// z = M^(-1)r
				rhOld = rhNew;
				DOTGPU<T,128><<<params.blocks,params.blockSize,params.blockSize*sizeof(T)>>>(drh, drx, dzx, dims.Nx-1, dims.Ny,dims.Nz);
				hipMemcpy(hrh, drh, params.blocks*sizeof(T), hipMemcpyDeviceToHost);
				rhNew = dot(hrh,params.blocks);
			}
			//cout << "Ux iter number: " << iter << endl;
			// ********** END solve UX ************
			
			// ********** BEGIN solve UY **********
			duToDr<<<BLOCKS,THREADS_PER_BLOCK>>>(dry,duy,dims.Nx,dims.Ny-1,dims.Nz);					// dry := duy
			SpMV<<<BLOCKS,THREADS_PER_BLOCK>>>(dqy,duyc,duyf,duys,duyw,dry,dims.Nx,dims.Ny-1,dims.Nz);	// q := Auy uy 
			//duToDr<<<BLOCKS,THREADS_PER_BLOCK>>>(dry, duyhalf,dims.Nx,dims.Ny-1,dims.Nz);
			duToDr<<<BLOCKS,THREADS_PER_BLOCK>>>(dry,duyo,dims.Nx,dims.Ny-1,dims.Nz);
			b<<<BLOCKS,THREADS_PER_BLOCK>>>(dry,dims.Nx,dims.Ny-1,dims.Nz);	
			//byOutlet not necessary due to zero gradient condition
			expMuy<<<BLOCKS,THREADS_PER_BLOCK>>>(dry,duyo,dims.Nx,dims.Ny-1,dims.Nz); // explicit part of diffusion term (Crank-Nicolson scheme)
			bpy<<<BLOCKS,THREADS_PER_BLOCK>>>(dry,dpo,dims.Nx,dims.Ny-1,dims.Nz);						// add grad(p) to rhs of Ax=b
			AXPY<<<BLOCKS,THREADS_PER_BLOCK>>>(dry,dqy,(T)-1.,(T)1.,dims.Nx,dims.Ny-1,dims.Nz);   		// r = r - q
			SpMV<<<BLOCKS,THREADS_PER_BLOCK>>>(dzy,dkuyc,dkuyf,dkuys,dkuyw,dry,dims.Nx,dims.Ny-1,dims.Nz);	// z = M^(-1)r
			DOTGPU<T,128><<<params.blocks,params.blockSize,params.blockSize*sizeof(T)>>>(drh, dry, dzy, dims.Nx, dims.Ny-1,dims.Nz);
			hipMemcpy(hrh, drh, params.blocks*sizeof(T), hipMemcpyDeviceToHost);
			rhNew = dot(hrh,params.blocks);
			//cout << "Uy residual at start: " << rhNew << endl;
			endIter = rhNew * params.maxResU * params.maxResU;
			iter = 0;
			
			while (rhNew > endIter) {
				iter++;
				if (iter==1) {
					hipMemcpy(dpy, dzy, sizeof(T)*dims.Nx*(dims.Ny-1)*(dims.Nz+2),hipMemcpyDeviceToDevice);
				}
				else {
					bt = rhNew/rhOld;
					AXPY<<<BLOCKS,THREADS_PER_BLOCK>>>(dpy,dzy,(T)1.,bt,dims.Nx,dims.Ny-1,dims.Nz);   		    // p = z + beta*p	
				}
				SpMV<<<BLOCKS,THREADS_PER_BLOCK>>>(dqy,duyc,duyf,duys,duyw,dpy,dims.Nx,dims.Ny-1,dims.Nz);		// q := Auy p
				DOTGPU<T,128><<<params.blocks,params.blockSize,params.blockSize*sizeof(T)>>>(dsg,dpy,dqy,dims.Nx,dims.Ny-1,dims.Nz);
				hipMemcpy(hsg, dsg, params.blocks*sizeof(T), hipMemcpyDeviceToHost);
				sg = dot(hsg,params.blocks);
				ap = rhNew/sg;	// alpha = rhoNew / sigma
				AXPY<<<BLOCKS,THREADS_PER_BLOCK>>>(dry,dqy,-ap,(T)1.,dims.Nx,dims.Ny-1,dims.Nz);  			    // r = r - alpha*q
				AXPY2<<<BLOCKS,THREADS_PER_BLOCK>>>(duy,dpy, ap,(T)1.,dims.Nx,dims.Ny-1,dims.Nz);  			    // x = x + alpha*p; Note: sizeof(duy) != sizeof(dpy)
				SpMV<<<BLOCKS,THREADS_PER_BLOCK>>>(dzy,dkuyc,dkuyf,dkuys,dkuyw,dry,dims.Nx,dims.Ny-1,dims.Nz);	// z = M^(-1)r
				rhOld = rhNew;
				DOTGPU<T,128><<<params.blocks,params.blockSize,params.blockSize*sizeof(T)>>>(drh, dry, dzy, dims.Nx, dims.Ny-1,dims.Nz);
				hipMemcpy(hrh, drh, params.blocks*sizeof(T), hipMemcpyDeviceToHost);
				rhNew = dot(hrh,params.blocks);
			}
			//cout << "Uy iter number: " << iter << endl;
			// ********** END solve UY ************
			
			
			// ********** BEGIN solve UZ **********
			duToDr<<<BLOCKS,THREADS_PER_BLOCK>>>(drz,duz,dims.Nx,dims.Ny,dims.Nz-1);					// dry := duy
			SpMV<<<BLOCKS,THREADS_PER_BLOCK>>>(dqz,duzc,duzf,duzs,duzw,drz,dims.Nx,dims.Ny,dims.Nz-1);	// q := Auy uy 
			//duToDr<<<BLOCKS,THREADS_PER_BLOCK>>>(drz, duzhalf,dims.Nx,dims.Ny,dims.Nz-1);
			duToDr<<<BLOCKS,THREADS_PER_BLOCK>>>(drz,duzo,dims.Nx,dims.Ny,dims.Nz-1);
			b<<<BLOCKS,THREADS_PER_BLOCK>>>(drz,dims.Nx,dims.Ny,dims.Nz-1);
			bzInlet<<<1,100>>>(drz, duz, 50, 45);
			//bzOutlet not necessary due to zero gradient condition
			expMuz<<<BLOCKS,THREADS_PER_BLOCK>>>(drz,duzo,dims.Nx,dims.Ny,dims.Nz-1); // explicit part of diffusion term (Crank-Nicolson scheme)
			bpz<<<BLOCKS,THREADS_PER_BLOCK>>>(drz,dpo,dims.Nx,dims.Ny,dims.Nz-1);						// add grad(p) to rhs of Ax=b
			AXPY<<<BLOCKS,THREADS_PER_BLOCK>>>(drz,dqz,(T)-1.,(T)1.,dims.Nx,dims.Ny,dims.Nz-1);   		// r = r - q
			SpMV<<<BLOCKS,THREADS_PER_BLOCK>>>(dzz,dkuzc,dkuzf,dkuzs,dkuzw,drz,dims.Nx,dims.Ny,dims.Nz-1);	// z = M^(-1)r
			DOTGPU<T,128><<<params.blocks,params.blockSize,params.blockSize*sizeof(T)>>>(drh, drz, dzz, dims.Nx, dims.Ny,dims.Nz-1);
			hipMemcpy(hrh, drh, params.blocks*sizeof(T), hipMemcpyDeviceToHost);
			rhNew = dot(hrh,params.blocks);
			//cout << "Uz residual at start: " << rhNew << endl;
			endIter = rhNew * params.maxResU * params.maxResU;
			iter = 0;
			
			while (rhNew > endIter) {
				iter++;
				if (iter==1) {
					hipMemcpy(dpz, dzz, sizeof(T)*dims.Nx*dims.Ny*(dims.Nz+1),hipMemcpyDeviceToDevice);
				}
				else {
					bt = rhNew/rhOld;
					AXPY<<<BLOCKS,THREADS_PER_BLOCK>>>(dpz,dzz,(T)1.,bt,dims.Nx,dims.Ny,dims.Nz-1);   		    // p = z + beta*p	
				}
				SpMV<<<BLOCKS,THREADS_PER_BLOCK>>>(dqz,duzc,duzf,duzs,duzw,dpz,dims.Nx,dims.Ny,dims.Nz-1);		// q := Auz p
				DOTGPU<T,128><<<params.blocks,params.blockSize,params.blockSize*sizeof(T)>>>(dsg,dpz,dqz,dims.Nx,dims.Ny,dims.Nz-1);
				hipMemcpy(hsg, dsg, params.blocks*sizeof(T), hipMemcpyDeviceToHost);
				sg = dot(hsg,params.blocks);
				ap = rhNew/sg;	// alpha = rhoNew / sigma
				AXPY<<<BLOCKS,THREADS_PER_BLOCK>>>(drz,dqz,-ap,(T)1.,dims.Nx,dims.Ny,dims.Nz-1);  			    // r = r - alpha*q
				AXPY2<<<BLOCKS,THREADS_PER_BLOCK>>>(duz,dpz, ap,(T)1.,dims.Nx,dims.Ny,dims.Nz-1);  			    // x = x + alpha*p; Note: sizeof(duz) != sizeof(dpz)
				SpMV<<<BLOCKS,THREADS_PER_BLOCK>>>(dzz,dkuzc,dkuzf,dkuzs,dkuzw,drz,dims.Nx,dims.Ny,dims.Nz-1);	// z = M^(-1)r
				rhOld = rhNew;
				DOTGPU<T,128><<<params.blocks,params.blockSize,params.blockSize*sizeof(T)>>>(drh, drz, dzz, dims.Nx, dims.Ny,dims.Nz-1);
				hipMemcpy(hrh, drh, params.blocks*sizeof(T), hipMemcpyDeviceToHost);
				rhNew = dot(hrh,params.blocks);


			}
			//cout << "Uz iter number: " << iter << endl;
			// ********** END solve UZ ************
			
				
			// update velocity at boundary
			bcVelOutlet<<<1,100>>>(dux, duy, duz, 200, 15);
			
			
			
			// ********** BEGIN solve P ***********
			// The finite volume method in computational fluid dynamics, F. Moukalled, L. Mangani, M. Darwish
			// Patankar's SIMPLE
			hipMemcpy(drp,dp,sizeof(T)*dims.Nx*dims.Ny*(dims.Nz+2),hipMemcpyDeviceToDevice);
			SpMV<<<BLOCKS,THREADS_PER_BLOCK>>>(dqp,dpc,dpf,dps,dpw,drp,dims.Nx,dims.Ny,dims.Nz);			// q := Ap p 
			bp<<<BLOCKS,THREADS_PER_BLOCK>>>(drp,dux,duy,duz,dims.Nx,dims.Ny,dims.Nz);					// should become at convergence == zero correction field

#ifdef DEFL // store rhs (b) for correction
			hipMemcpy(drhs,drp,sizeof(T)*dims.Nx*dims.Ny*(dims.Nz+2),hipMemcpyDeviceToDevice);
#endif
			
			DOTGPU<T,128><<<params.blocks,params.blockSize,params.blockSize*sizeof(T)>>>(drh,drp,drp,dims.Nx,dims.Ny,dims.Nz);
			hipMemcpy(hrh, drh, params.blocks*sizeof(T), hipMemcpyDeviceToHost);
			rhNewSIMPLE = dot(hrh,params.blocks);
						
			AXPY<<<BLOCKS,THREADS_PER_BLOCK>>>(drp,dqp,(T)-1.,(T)1.,dims.Nx,dims.Ny,dims.Nz);   			// r = r - q
			
#ifdef DEFL	 
			localDOTGPU<T,256><<<256,dimBlockZ,256*sizeof(T)>>>(drZ,drp); // equivalent to ZTransXYDeflation
			hipMemcpy(hrZ,drZ,(paramsZ.nDV+2*paramsZ.NxZ*paramsZ.NyZ)*sizeof(T),hipMemcpyDeviceToHost);   // copy drZ to hrZ
#ifdef DEFLDIR
			solveDC(hyZ,hrZ,L);
#else
			solveICCG(hsZ,hrZ,hyZ,hpZ,hqZ,
					ec,ef,es,ew,
					lc,lf,ls,lw);
#endif
			hipMemcpy(dyZ,hyZ,(paramsZ.nDV+2*paramsZ.NxZ*paramsZ.NyZ)*sizeof(T),hipMemcpyHostToDevice);   //copy hyZ to dyZ
			YMinusAzXYDeflation<<<BLOCKS,THREADS_PER_BLOCK>>>(drp,dyZ,dpzc,dpzf,dpzs,dpzw);  // r = P*r
			
				
			
			//cout << "stopped" << endl;
			//break;
#endif 
			
			SpMV<<<BLOCKS,THREADS_PER_BLOCK>>>(dzp,dkpc,dkpf,dkps,dkpw,drp,dims.Nx,dims.Ny,dims.Nz);		// z = M^(-1)r
			
			
			DOTGPU<T,128><<<params.blocks,params.blockSize,params.blockSize*sizeof(T)>>>(drh,drp,dzp,dims.Nx,dims.Ny,dims.Nz);
			hipMemcpy(hrh, drh, params.blocks*sizeof(T), hipMemcpyDeviceToHost);
			rhNew = dot(hrh,params.blocks);
			//cout << "P residual at start: " << rhNew << endl;
			
			if (iterSIMPLE==1)	endIterP = rhNew * params.maxResP * params.maxResP;
						
			iter = 0;
			
			while (rhNew > endIterP) {  //(iter<8) {
				iter++;
				//cout << "iteration:" << iter << ", residual: " << setprecision(11) << rhNew << endl;
				if (iter==1) {
					hipMemcpy(dpp, dzp, sizeof(T)*dims.Nx*dims.Ny*(dims.Nz+2),hipMemcpyDeviceToDevice);
				}
				else {
					bt = rhNew/rhOld;
					AXPY<<<BLOCKS,THREADS_PER_BLOCK>>>(dpp,dzp,(T)1.,bt,dims.Nx,dims.Ny,dims.Nz);   		// p = z + beta*p	
				}
				SpMV<<<BLOCKS,THREADS_PER_BLOCK>>>(dqp,dpc,dpf,dps,dpw,dpp,dims.Nx,dims.Ny,dims.Nz);		// q := Ap p
#ifdef DEFL	 
				localDOTGPU<T,256><<<256,dimBlockZ,256*sizeof(T)>>>(drZ,dqp); // equivalent to ZTransXYDeflation
				hipMemcpy(hrZ,drZ,(paramsZ.nDV+2*paramsZ.NxZ*paramsZ.NyZ)*sizeof(T),hipMemcpyDeviceToHost);   // copy drZ to hrZ
#ifdef DEFLDIR
				solveDC(hyZ,hrZ,L);
#else
				solveICCG(hsZ,hrZ,hyZ,hpZ,hqZ,
						ec,ef,es,ew,
						lc,lf,ls,lw);
#endif
				hipMemcpy(dyZ,hyZ,(paramsZ.nDV+2*paramsZ.NxZ*paramsZ.NyZ)*sizeof(T),hipMemcpyHostToDevice);   //copy hyZ to dyZ
				YMinusAzXYDeflation<<<BLOCKS,THREADS_PER_BLOCK>>>(dqp,dyZ,dpzc,dpzf,dpzs,dpzw);  // r = P*r
#endif 
		        DOTGPU<T,128><<<params.blocks,params.blockSize,params.blockSize*sizeof(T)>>>(dsg,dpp,dqp,dims.Nx,dims.Ny,dims.Nz);
				hipMemcpy(hsg, dsg, params.blocks*sizeof(T), hipMemcpyDeviceToHost);
				sg = dot(hsg,params.blocks);
				ap = rhNew/sg;	// alpha = rhoNew / sigma
				AXPY<<<BLOCKS,THREADS_PER_BLOCK>>>(drp,dqp,-ap,(T)1.,dims.Nx,dims.Ny,dims.Nz);  			// r = r - alpha*q
				AXPY<<<BLOCKS,THREADS_PER_BLOCK>>>(dp ,dpp, ap,(T)1.,dims.Nx,dims.Ny,dims.Nz);  			// x = x + alpha*p
				SpMV<<<BLOCKS,THREADS_PER_BLOCK>>>(dzp,dkpc,dkpf,dkps,dkpw,drp,dims.Nx,dims.Ny,dims.Nz);	// z = M^(-1)r
				rhOld = rhNew;
				DOTGPU<T,128><<<params.blocks,params.blockSize,params.blockSize*sizeof(T)>>>(drh,drp,dzp,dims.Nx,dims.Ny,dims.Nz);
				hipMemcpy(hrh, drh, params.blocks*sizeof(T), hipMemcpyDeviceToHost);
				rhNew = dot(hrh,params.blocks);
			}
#ifdef DEFL	 // y:= Q*b + P^T*y
			// P^T * y
			    SpMV<<<BLOCKS,THREADS_PER_BLOCK>>>(dqp,dpc,dpf,dps,dpw,dp,dims.Nx,dims.Ny,dims.Nz);
				localDOTGPU<T,256><<<256,dimBlockZ,256*sizeof(T)>>>(drZ,dqp); // equivalent to ZTransXYDeflation
				hipMemcpy(hrZ,drZ,(paramsZ.nDV+2*paramsZ.NxZ*paramsZ.NyZ)*sizeof(T),hipMemcpyDeviceToHost);   // copy drZ to hrZ
#ifdef DEFLDIR
				solveDC(hyZ,hrZ,L);
#else
				solveICCG(hsZ,hrZ,hyZ,hpZ,hqZ,
						ec,ef,es,ew,
						lc,lf,ls,lw);
#endif
				hipMemcpy(dyZ,hyZ,(paramsZ.nDV+2*paramsZ.NxZ*paramsZ.NyZ)*sizeof(T),hipMemcpyHostToDevice);   //copy hyZ to dyZ  (= y2)
				YMinusZXYDeflation<<<BLOCKS,THREADS_PER_BLOCK>>>(dp,dyZ);  // P^T*y := y -Z*y2
				localDOTGPU<T,256><<<256,dimBlockZ,256*sizeof(T)>>>(drZ,drhs); // equivalent to ZTransXYDeflation
				hipMemcpy(hrZ,drZ,(paramsZ.nDV+2*paramsZ.NxZ*paramsZ.NyZ)*sizeof(T),hipMemcpyDeviceToHost);   // copy drZ to hrZ
#ifdef DEFLDIR
				solveDC(hyZ,hrZ,L);
#else
				solveICCG(hsZ,hrZ,hyZ,hpZ,hqZ,
						ec,ef,es,ew,
						lc,lf,ls,lw);
#endif
				hipMemcpy(dyZ,hyZ,(paramsZ.nDV+2*paramsZ.NxZ*paramsZ.NyZ)*sizeof(T),hipMemcpyHostToDevice);   // copy hyZ to dyZ  (= y2)
				YPlusZXYDeflation<<<BLOCKS,THREADS_PER_BLOCK>>>(dp,dyZ);           // P^T*y + Q*b := y + Z*y2	
#endif 
			//cout << "P iter number: " << iter << endl;
			//cout << "P residual at end: " << rhNew << endl;
			// ********** END solve P ************
			
			
			// ***** BEGIN correct P, UX, UY fields ******
			correctUX<<<BLOCKS,THREADS_PER_BLOCK>>>(dux,dp,dims.Nx-1,dims.Ny,dims.Nz); 				// ux = -dt/rho*dp/dx
			correctUY<<<BLOCKS,THREADS_PER_BLOCK>>>(duy,dp,dims.Nx,dims.Ny-1,dims.Nz);				// uy = -dt/rho*dp/dy
			correctUZ<<<BLOCKS,THREADS_PER_BLOCK>>>(duz,dp,dims.Nx,dims.Ny,dims.Nz-1);				// uz = -dt/rho*dp/dz
			AXPY<<<BLOCKS,THREADS_PER_BLOCK>>>(dp,dpo,(T)1.,params.urfP,dims.Nx,dims.Ny,dims.Nz);	// p = urfP*p + pold
			hipMemcpy(dpo, dp, sizeof(T)*dims.Nx*dims.Ny*(dims.Nz+2),hipMemcpyDeviceToDevice);	// pold = p
			hipMemset(dp ,  0, sizeof(T)*dims.Nx*dims.Ny*(dims.Nz+2));
			
			bcVelOutlet<<<1,100>>>(dux, duy, duz, 200, 15);
			// ****** END correct P, UX, UY fields *******
			
			
			
			/*// ***** BEGIN check mass conservation *****
			bp<<<BLOCKS,THREADS_PER_BLOCK>>>(dm,dux,duy,duz,dims.Nx,dims.Ny,dims.Nz);
			hipMemcpy(m, dm, sizeof(T)*dims.Nx*dims.Ny*(dims.Nz+2), hipMemcpyDeviceToHost);
			// ****** END check mass conservation *******/
			
			
			/*// copy back to host and save
			hipMemcpy(ux, dux, sizeof(T)*(dims.Nx+2)*(dims.Ny+2)*(dims.Nz+2), hipMemcpyDeviceToHost);
			hipMemcpy(uy, duy, sizeof(T)*(dims.Nx+2)*(dims.Ny+2)*(dims.Nz+2), hipMemcpyDeviceToHost);
			hipMemcpy(uz, duz, sizeof(T)*(dims.Nx+2)*(dims.Ny+2)*(dims.Nz+2), hipMemcpyDeviceToHost);
			hipMemcpy(p,  dpo, sizeof(T)*dims.Nx    * dims.Ny   *(dims.Nz+2), hipMemcpyDeviceToHost);
			saveDataInTime(ux, uy, uz, p, m, (T)iterSIMPLE, "testTundish");*/
			
			
			
		
		}
		// ************** END SIMPLE *****************
		
		if (miter%10 == 0) {
		
		// ***** BEGIN check mass conservation *****
		bp<<<BLOCKS,THREADS_PER_BLOCK>>>(dm,dux,duy,duz,dims.Nx,dims.Ny,dims.Nz);
		hipMemcpy(m, dm, sizeof(T)*dims.Nx*dims.Ny*(dims.Nz+2), hipMemcpyDeviceToHost);
		// ****** END check mass conservation *******/
		
		// copy back to host and save
		hipMemcpy(ux, dux, sizeof(T)*(dims.Nx+2)*(dims.Ny+2)*(dims.Nz+2), hipMemcpyDeviceToHost);
		hipMemcpy(uy, duy, sizeof(T)*(dims.Nx+2)*(dims.Ny+2)*(dims.Nz+2), hipMemcpyDeviceToHost);
		hipMemcpy(uz, duz, sizeof(T)*(dims.Nx+2)*(dims.Ny+2)*(dims.Nz+2), hipMemcpyDeviceToHost);
		hipMemcpy(p,  dpo, sizeof(T)*dims.Nx    * dims.Ny   *(dims.Nz+2), hipMemcpyDeviceToHost);
		saveDataInTime(ux, uy, uz, p, m, (T)miter, "3Dtundish_LTcubic_mu1e-6Pas_CNdiff_CFL1_res1e-6");
		
		}
		
		
		cout << "SIMPLE iter number: " << iterSIMPLE << endl;
	}
	cout << "simulation finished." << endl;
	
	
	
	
	
	
	
	
	
	
	
	
	
	
		
	
	
	/*
	
	//hipMemcpy(p, drp, sizeof(T)*(dims.Nx)*(dims.Ny+2), hipMemcpyDeviceToHost);
				ofstream File;
				File.open("ckeck_pw");
				for (int i=0;i<dims.Nx*dims.Ny*dims.Nz;i++) {
						File << pw[i] << endl;
				}
				File.close();*/
	
	
	
	
	
	
		
	
	
	
	
	
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	cout<< "ellapsed time (cuda): " << elapsedTime	<< " miliseconds" << endl;
	
	/*// copy back to host and save
	hipMemcpy(ux, dux, sizeof(T)*(dims.Nx+2)*(dims.Ny+2)*(dims.Nz+2), hipMemcpyDeviceToHost);
	hipMemcpy(uy, duy, sizeof(T)*(dims.Nx+2)*(dims.Ny+2)*(dims.Nz+2), hipMemcpyDeviceToHost);
	hipMemcpy(uz, duz, sizeof(T)*(dims.Nx+2)*(dims.Ny+2)*(dims.Nz+2), hipMemcpyDeviceToHost);
	hipMemcpy(p,  dpo, sizeof(T)*dims.Nx    * dims.Ny   *(dims.Nz+2), hipMemcpyDeviceToHost);
	saveDataInTime(ux, uy, uz, p, m, (T)100, "testTundish-max20-all1e-2");*/
	
	
			
	cpuFinalize(ux, uy, uz, p, m, hrh, hsg);
	cudaFinalizeTricubicDerivatives(duxdx, duxdy, duxdz,
			duxdxdy, duxdxdz, duxdydz, duxdxdydz,
			duydx, duydy, duydz,
			duydxdy, duydxdz, duydydz, duydxdydz,
			duzdx, duzdy, duzdz,
			duzdxdy, duzdxdz, duzdydz, duzdxdydz,
			x_ux, y_ux, z_ux,
			x_uy, y_uy, z_uy,
			x_uz, y_uz, z_uz);
	cudaFinalize(dux, duy, duz, dp, dm, duxo, duyo, duzo, dpo,
			     duxhalf, duyhalf, duzhalf,
				 duxc, duxf, duxs, duxw, dkuxc, dkuxf, dkuxs, dkuxw, 	// Aux
				 drx, dqx, dzx, dpx,									// Aux
				 duyc, duyf, duys, duyw, dkuyc, dkuyf, dkuys, dkuyw, 	// Auy
				 dry, dqy, dzy, dpy,									// Auy
				 duzc, duzf, duzs, duzw, dkuzc, dkuzf, dkuzs, dkuzw, 	// Auz
				 drz, dqz, dzz, dpz,									// Auz
				 dpc, dpf, dps, dpw, dkpc, dkpf, dkps, dkpw, 			// Ap
				 drp, dqp, dzp, dpp,									// Ap
				 drh, dsg);
#ifdef DEFL
	cpuFinalizeDeflation(pzc, pzf, pzs, pzw,
			ec, ef, es, ew,
			pc, pf, ps, pw,
			lc, lf, ls, lw,
			hrZ,hyZ,hqZ,hpZ,hsZ,
			L);
	cudaFinalizeDeflation(dpzc,dpzf,dpzs,dpzw,
				dec,def,des,dew,
				drZ,dyZ,dqZ,dpZ,
				drhs);
#endif	
	
	return 0;
}
